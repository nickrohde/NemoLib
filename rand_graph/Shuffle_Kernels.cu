#pragma once

#ifndef __SHUFFLE_KERNELS_CUH
#define __SHUFFLE_KERNELS_CUH

#include <cstddef>
#include <hip/hip_runtime.h>
#include <iostream>
#include <string.h>


#include "hiprand/hiprand_kernel.h"

#ifndef CUDA_CALL
#define CUDA_CALL(x) do { auto y = (x); if(y != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
	std::cout << "Error is: " << hipGetErrorString(y) << std::endl; \
    exit(EXIT_FAILURE);}} while(0)
#endif

typedef unsigned int uint;


__global__ void setup_kernel(hiprandState *state)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	/* Each thread gets same seed, a different sequence
	number, no offset */
	hiprand_init(clock64(), id, 0, &state[id]);
}


template <typename T>
__device__ void shuffle_vector(T* __restrict vect, const std::size_t LENGTH, const std::size_t idx_2, hiprandState *state)
{
	std::size_t id = threadIdx.x + blockIdx.x * blockDim.x;
	T* my_v = &vect[id * LENGTH];

	float rand_num = hiprand_uniform(&state[id]);

	// random index for swap in range [idx_2, last]
	// simply picking a number in range [0, last] will
	// create bias in the shuffled list
	rand_num *= ((LENGTH - 1) - idx_2) + 0.999999;
	rand_num += idx_2;

	std::size_t idx_1 = (std::size_t)truncf(rand_num);

	// swap current element and random element
	T temp = my_v[idx_1];
	my_v[idx_1] = my_v[idx_2];
	my_v[idx_2] = temp;
}


template <typename T>
__global__ void duplicate_n(T* __restrict vect, const std::size_t LENGTH, const std::size_t N)
{
	std::size_t id = threadIdx.x + blockIdx.x * blockDim.x;
	if (id < LENGTH)
	{
		for (auto i = 1; i < N; i++)
		{
			std::size_t idx = id + i * LENGTH;
			vect[idx] = vect[id];
		}
	}
}


template <typename T>
__global__ void shuffle(T* __restrict vect, const std::size_t LENGTH, hiprandState* state)
{
	for (std::size_t i = 0; i < LENGTH; i++)
	{
		shuffle_vector(vect, LENGTH, i, state);
	}
}


 void shuffle_wrapper(uint n_blocks, uint n_threads, hipStream_t& stream, int* __restrict vect, const std::size_t LENGTH, hiprandState* state)
{
	shuffle<int><<<n_blocks, n_threads, 0, stream>>>(vect, LENGTH, state);
} 


 void duplicate_n_wrapper(uint n_blocks, uint n_threads, hipStream_t& stream, int* __restrict vect, const std::size_t LENGTH, const std::size_t N)
{
	duplicate_n<int><<<n_blocks, n_threads, 0, stream>>>(vect, LENGTH, N);
}


 void setup_kernel_wrapper(uint n_blocks, uint n_threads, hipStream_t& stream, hiprandState *state)
{
	setup_kernel<<<n_blocks, n_threads, 0, stream>>>(state);
}

#endif